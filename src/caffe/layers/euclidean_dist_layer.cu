#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/common_layers.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void EuclideanDistLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  int count = bottom[0]->count();
  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),  // a
      bottom[1]->gpu_data(),  // b
      diff_.mutable_gpu_data());  // diff_i=a_i-b_i
  const int channels = bottom[0]->channels();
  //Dtype margin = this->layer_param_.euclidean_dist_param().margin();
  //Dtype loss(0.0);
  for (int i = 0; i < bottom[0]->num(); ++i) {
    Dtype dot;
    caffe_gpu_dot(channels,
        diff_.gpu_data() + (i*channels), diff_.gpu_data() + (i*channels), &dot);
    (*top)[0]->mutable_cpu_data()[i]=dot; 
    //dist_sq_.mutable_cpu_data()[i] = caffe_cpu_dot(channels,
        //diff_.cpu_data() + (i*channels), diff_.cpu_data() + (i*channels));
    //if (static_cast<int>(bottom[2]->cpu_data()[i])) {  // similar pairs
    //  loss += dist_sq_.cpu_data()[i];
    //} else {  // dissimilar pairs
    //  loss += std::max(margin-dist_sq_.cpu_data()[i], Dtype(0.0));
    //}
  }
  //loss = loss / static_cast<Dtype>(bottom[0]->num()) / Dtype(2);
  //(*top)[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
void EuclideanDistLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, vector<Blob<Dtype>*>* bottom) {
  NOT_IMPLEMENTED;
}

INSTANTIATE_CLASS(EuclideanDistLayer);

}  // namespace caffe
